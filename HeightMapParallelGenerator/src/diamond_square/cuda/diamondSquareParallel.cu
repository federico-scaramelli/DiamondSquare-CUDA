#include "hip/hip_runtime.h"
﻿#include "diamondSquareParallel.h"

#pragma region CheckCUDACalls

#define CHECK_CURAND(call)                                                     \
{                                                                              \
    hiprandStatus_t err;                                                        \
    if ((err = (call)) != HIPRAND_STATUS_SUCCESS)                               \
    {                                                                          \
        fprintf(stderr, "Got CURAND error %d at %s:%d\n", err, __FILE__,       \
                __LINE__);                                                     \
        exit(1);                                                               \
    }                                                                          \
}

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
    }                                                                          \
}

#pragma endregion

#pragma region Random Generator

#include "hiprand.h"

void DiamondSquareParallel::PrintRandoms ()
{
	/*randoms = new float[totalSize];
	CHECK(hipMemcpy(randoms, dev_Randoms, totalSize * sizeof(float), hipMemcpyDeviceToHost))
	auto count = 0;
	/* Show result #1#
	for (int i = 0; i < totalSize; i++) {
		count = getRandom(randoms + i) ? count + 1 : count;
		//std::cout << randoms[i];
	}
	std::cout << count << " negativi" << std::endl;
	std::cout << totalSize - count << " positivi" << std::endl;
	delete[] randoms;*/
}

void DiamondSquareParallel::GenerateRandomNumbers ()
{
	int seed = random_int_uniform();
	hiprandGenerator_t generator;
	CHECK_CURAND (hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MT19937))
	CHECK_CURAND (hiprandSetPseudoRandomGeneratorSeed(generator, seed))

	/* Allocate n floats on device */
	CHECK (hipMalloc((void **)&dev_Randoms, totalSize * sizeof(float)))

	/* Generate n floats on device */
	CHECK_CURAND (hiprandGenerateUniform(generator, dev_Randoms, totalSize))

	//PrintRandoms();

	/* Cleanup */
	CHECK_CURAND (hiprandDestroyGenerator(generator))
}

/* TODO: change it to void */
inline void DiamondSquareParallel::getRandom (float* const value)
{
	bool cond = static_cast<int> (*value * 10) / 1 & 0x01;
	*value = *value * (-1) * cond + *value * !cond;
}

__device__ __forceinline__ float getRandomOnDevice (float const value)
{
	bool cond = static_cast<int> (value * 10) / 1 & 0x01;
	return value * (-1) * cond + value * !cond;
}

#pragma endregion


void DiamondSquareParallel::InitializeDiamondSquare ()
{
	MeasureTimeFn ("Parallel random numbers generation time: ", this, &DiamondSquareParallel::GenerateRandomNumbers);

	/* For now initialize on the CPU side
	 * TODO: initialize values on the GPU */
	for (uint32_t x = 0; x < size; x += step) {
		for (uint32_t y = 0; y < size; y += step) {
			map[GetIndex (x, y)] = random_float_uniform();
		}
	}

	MeasureTimeFn ("Copy initial map to the device time: ", this, &DiamondSquareParallel::CopyMapToDevice);
}

void DiamondSquareParallel::CopyMapToDevice ()
{
	/* Copy the map on the device memory */
	CHECK (hipMalloc(&dev_Map, totalSize * sizeof(float)))
	CHECK (hipMemcpy(dev_Map, map, totalSize * sizeof(float), hipMemcpyHostToDevice))
}

void DiamondSquareParallel::DiamondSquare ()
{
	while (step > 1) {
		DiamondStep();
		hipDeviceSynchronize();
		CHECK (hipMemcpy(map, dev_Map, totalSize * sizeof(float), hipMemcpyDeviceToHost))
		PrintMap();

		SquareStep();
		hipDeviceSynchronize();
		CHECK (hipMemcpy(map, dev_Map, totalSize * sizeof(float), hipMemcpyDeviceToHost))
		PrintMap();

		randomScale /= 2.0f;
		step /= 2;
		blockSizeDiamond *= 2;
		blockSizeSquare *= 2;
	}

	CHECK (hipMemcpy(map, dev_Map, totalSize * sizeof(float), hipMemcpyDeviceToHost))

	CleanUp();
}

__device__ __forceinline__ uint32_t GetIndex (uint32_t x, uint32_t y, uint32_t size)
{
	x = x >= size ? size - 1 : x;
	y = y >= size ? size - 1 : y;

	return x * size + y;
}

__global__ void DiamondStepParallel (float* map, float* randoms, uint32_t size, uint32_t step, float randomScale)
{
	uint32_t half = step / 2;

	uint32_t thd_X = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t thd_Y = blockIdx.x * blockDim.x + threadIdx.x;
	thd_X = thd_X * step + half;
	thd_Y = thd_Y * step + half;

	float val = map[GetIndex (thd_X - half, thd_Y - half, size)] +
		map[GetIndex (thd_X + half, thd_Y - half, size)] +
		map[GetIndex (thd_X - half, thd_Y + half, size)] +
		map[GetIndex (thd_X + half, thd_Y + half, size)];

	val /= 4.0f;
	val += randomScale * getRandomOnDevice(randoms[GetIndex (thd_X, thd_Y, size)]);

	map[GetIndex (thd_X, thd_Y, size)] = val;
}

void DiamondSquareParallel::DiamondStep ()
{
	dim3 blockDimension (blockSizeDiamond, blockSizeDiamond, 1);
	dim3 gridDimension (1, 1, 1);
	DiamondStepParallel<<<gridDimension, blockDimension>>> (dev_Map, dev_Randoms, size, step, randomScale);
}

__global__ void SquareStepParallel (float* map, float* randoms, uint32_t size, uint32_t step, float randomScale)
{
	uint32_t half = step / 2;

	uint32_t thd_X = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t thd_Y = blockIdx.y * blockDim.y + threadIdx.y;

	uint32_t x = thd_X * step * (thd_Y % 2 == 0) +
		thd_Y * half * (thd_Y % 2 != 0);
	uint32_t y = (thd_Y * half + half) * (thd_Y % 2 == 0) +
		thd_X * step * (thd_Y % 2 != 0);
	
	float val = map[GetIndex (x - half, y, size)] +
		map[GetIndex (x + half, y, size)] +
		map[GetIndex (x, y - half, size)] +
		map[GetIndex (x, y + half, size)];

	val /= 4.0f;
	val += randomScale * getRandomOnDevice(randoms[GetIndex (x, y, size)]);

	map[GetIndex (x, y, size)] = val;
}

void DiamondSquareParallel::SquareStep ()
{
	dim3 blockDimension (blockSizeDiamond + 1, blockSizeSquare, 1);
	std::cout << "Square step (" << blockSizeSquare << ", " << blockSizeDiamond + 1 << ");\n";
	dim3 gridDimension (1, 1, 1);
	SquareStepParallel<<<gridDimension, blockDimension>>> (dev_Map, dev_Randoms, size, step, randomScale);
}

void DiamondSquareParallel::CleanUp ()
{
	CHECK (hipFree(dev_Randoms))
	CHECK (hipFree(dev_Map))
}


__global__ void InitializeDiamondSquareParallel ()
{ }
