#include "hip/hip_runtime.h"
﻿#include "diamondSquareParallel.h"
#include "../parameters/applicationSettings.h"

#pragma region CheckCUDACalls

#define CHECK_CURAND(call)                                                     \
{                                                                              \
    hiprandStatus_t err;                                                        \
    if ((err = (call)) != HIPRAND_STATUS_SUCCESS)                               \
    {                                                                          \
        fprintf(stderr, "Got CURAND error %d at %s:%d\n", err, __FILE__,       \
                __LINE__);                                                     \
        exit(1);                                                               \
    }                                                                          \
}

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
    }                                                                          \
}

#pragma endregion

#pragma region Random Generator

#include "hiprand.h"

void DiamondSquareParallel::PrintRandoms ()
{
	/*randoms = new float[totalSize];
	CHECK(hipMemcpy(randoms, dev_Randoms, totalSize * sizeof(float), hipMemcpyDeviceToHost))
	auto count = 0;
	/* Show result #1#
	for (int i = 0; i < totalSize; i++) {
		count = getRandom(randoms + i) ? count + 1 : count;
		//std::cout << randoms[i];
	}
	std::cout << count << " negativi" << std::endl;
	std::cout << totalSize - count << " positivi" << std::endl;
	delete[] randoms;*/
}

void DiamondSquareParallel::GenerateRandomNumbers ()
{
	int seed = RandomIntUniform();
	hiprandGenerator_t generator;
	CHECK_CURAND (hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MT19937))
	CHECK_CURAND (hiprandSetPseudoRandomGeneratorSeed(generator, seed))

	/* Allocate n floats on device */
	CHECK (hipMalloc((void **)&dev_Randoms, totalSize * sizeof(float)))

	/* Generate n floats on device */
	CHECK_CURAND (hiprandGenerateUniform(generator, dev_Randoms, totalSize))

	//PrintRandoms();

	/* Cleanup */
	CHECK_CURAND (hiprandDestroyGenerator(generator))
}

/* TODO: change it to void */
inline void DiamondSquareParallel::getRandom (float* const value)
{
	bool cond = static_cast<int> (*value * 10) / 1 & 0x01;
	*value = *value * (-1) * cond + *value * !cond;
}

__device__ __forceinline__ float getRandomOnDevice (float const value)
{
	bool cond = static_cast<int> (value * 10) / 1 & 0x01;
	return value * (-1) * cond + value * !cond;
}

#pragma endregion

__device__ __forceinline__ uint32_t GetIndex (uint32_t x, uint32_t y, uint32_t size)
{
	x = x >= size ? size - 1 : x;
	y = y >= size ? size - 1 : y;

	return x * size + y;
}

__global__ void InitializeDiamondSquareParallel (float* map, const float* randoms, uint32_t size, uint32_t step)
{
	uint32_t thd_X = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t thd_Y = blockIdx.x * blockDim.x + threadIdx.x;
	thd_X *= step;
	thd_Y *= step;

	map[GetIndex (thd_X, thd_Y, size)] = randoms[GetIndex (thd_X, thd_Y, size)]; 
}

void DiamondSquareParallel::InitializeDiamondSquare ()
{
  	std::cout << "==================PARALLEL DIAMOND SQUARE==================" << std::endl << std::endl;
  	std::cout << "----------INITIALIZATION----------" << std::endl;
	std::cout << "Initializing Diamond Square [" << size << " x " << size << "]..." << std::endl;
	MeasureTimeFn (nullptr, "Random number set generated in ", this, &DiamondSquareParallel::GenerateRandomNumbers);
	MeasureTimeFn (nullptr, "Initial map copied in the device memory in ", this, &DiamondSquareParallel::CopyMapToDevice);
	
	threadAmount = (size - 1) / step;
	uint32_t blockSize = threadAmount <= 16 ? threadAmount : 16;
	uint32_t gridSize = threadAmount / 16;
	dim3 blockDimension (blockSize, blockSize, 1);
	//std::cout << "Diamond step block size (" << blockSizeDiamond << ", " << blockSizeDiamond << ");\n";
	dim3 gridDimension (gridSize, gridSize, 1);
	//std::cout << "Thread amount: " << threadAmount << ", blockSize: " << blockSize << ", gridSize: " << gridSize;
	//std::cout << "Diamond step grid size (" << gridSizeDiamond << ", " << gridSizeDiamond << ");\n";
	InitializeDiamondSquareParallel<<<gridDimension, blockDimension>>> (dev_Map, dev_Randoms, size, step);

	/* For now initialize on the CPU side
	 * TODO: initialize values on the GPU */
	/*for (uint32_t x = 0; x < size; x += step) {
		for (uint32_t y = 0; y < size; y += step) {
			map[GetIndex (x, y)] = RandomFloatUniform();
		}
	}
	MeasureTimeFn ("Copy initial map to the device time: ", this, &DiamondSquareParallel::CopyMapToDevice);
	*/

}

void DiamondSquareParallel::CopyMapToDevice ()
{
	/* Copy the map on the device memory */
	CHECK (hipMalloc(&dev_Map, totalSize * sizeof(float)))
	CHECK (hipMemcpy(dev_Map, map, totalSize * sizeof(float), hipMemcpyHostToDevice))
}

void DiamondSquareParallel::CalculateBlockGridSizes ()
{
	/*			  2^k			  or			  16			  */
	blockSizeDiamond = threadAmount <= 16 ? threadAmount : 16;
	/*		(2^k + 1) x 2^(k+1)	  or			 8 x 16
	*		        k <= 3					     k > 3			  */
	blockXSizeSquare = threadAmount <= 8 ? blockSizeDiamond + 1 : 8;
	blockYSizeSquare = threadAmount <= 8 ? threadAmount * 2 : blockSizeDiamond;

	/*				  1			  or			2^k / 16		  */
	gridSizeDiamond = threadAmount <= 8 ? 1 : threadAmount / 16;
	/* 9 x 16 block amount =  (2^k / 16)  /	  ceil(2^k / 8)		  */
	gridSizeSquare = threadAmount < 16 ? 1 : (threadAmount + 8) / 8;
}

void DiamondSquareParallel::DiamondSquare ()
{
#if CUDA_EVENTS_TIMING
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord( start, 0 );
#endif

	while (step > 1) {
		CalculateBlockGridSizes();

		DiamondStep();
		hipDeviceSynchronize();
#if PRINT_DIAMOND_STEP_CUDA
		CHECK (hipMemcpy(map, dev_Map, totalSize * sizeof(float), hipMemcpyDeviceToHost))
		PrintMap();
#endif

		SquareStep();
		hipDeviceSynchronize();
#if PRINT_SQUARE_STEP_CUDA
		CHECK (hipMemcpy(map, dev_Map, totalSize * sizeof(float), hipMemcpyDeviceToHost))
		PrintMap();
#endif

		randomScale /= 2.0f;
		step /= 2;

		/* 2^k */
		threadAmount *= 2;
	}

#if COPY_RESULT_ON_HOST
	CHECK (hipMemcpy(map, dev_Map, totalSize * sizeof(float), hipMemcpyDeviceToHost))
#endif

	CleanUp();

#if CUDA_EVENTS_TIMING
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );

	hipEventElapsedTime( &executionTimeCuda, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );
#endif
}

__global__ void DiamondStepParallel (float* map, float* randoms, uint32_t size, uint32_t step, float randomScale)
{
	uint32_t half = step / 2;

	uint32_t thd_X = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t thd_Y = blockIdx.x * blockDim.x + threadIdx.x;
	thd_X = thd_X * step + half;
	thd_Y = thd_Y * step + half;

	float val = map[GetIndex (thd_X - half, thd_Y - half, size)] +
		map[GetIndex (thd_X + half, thd_Y - half, size)] +
		map[GetIndex (thd_X - half, thd_Y + half, size)] +
		map[GetIndex (thd_X + half, thd_Y + half, size)];

	val /= 4.0f;
	val += randomScale * getRandomOnDevice(randoms[GetIndex (thd_X, thd_Y, size)]);

	map[GetIndex (thd_X, thd_Y, size)] = val;
}

void DiamondSquareParallel::DiamondStep ()
{
	dim3 blockDimension (blockSizeDiamond, blockSizeDiamond, 1);
	//std::cout << "Diamond step block size (" << blockSizeDiamond << ", " << blockSizeDiamond << ");\n";
	dim3 gridDimension (gridSizeDiamond, gridSizeDiamond, 1);
	//std::cout << "Diamond step grid size (" << gridSizeDiamond << ", " << gridSizeDiamond << ");\n";
	DiamondStepParallel<<<gridDimension, blockDimension>>> (dev_Map, dev_Randoms, size, step, randomScale);
}

__global__ void SquareStepParallel (float* map, float* randoms, uint32_t size, uint32_t step, float randomScale)
{
	uint32_t half = step / 2;

	uint32_t thd_X = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t thd_Y = blockIdx.y * blockDim.y + threadIdx.y;

	uint32_t x = thd_X * step * (thd_Y % 2 == 0) +
		thd_Y * half * (thd_Y % 2 != 0);
	uint32_t y = (thd_Y * half + half) * (thd_Y % 2 == 0) +
		thd_X * step * (thd_Y % 2 != 0);

	if (x > size || y > size) {
		return;
	}
	
	float val = map[GetIndex (x - half, y, size)] +
		map[GetIndex (x + half, y, size)] +
		map[GetIndex (x, y - half, size)] +
		map[GetIndex (x, y + half, size)];

	val /= 4.0f;
	val += randomScale * getRandomOnDevice(randoms[GetIndex (x, y, size)]);

	map[GetIndex (x, y, size)] = val;
}

void DiamondSquareParallel::SquareStep ()
{
	dim3 blockDimension (blockXSizeSquare, blockYSizeSquare, 1);
	//std::cout << "Square step block size (" << blockXSizeSquare << ", " << blockYSizeSquare << ");\n";
	dim3 gridDimension (gridSizeSquare, threadAmount * 2 / 16, 1);
	//std::cout << "Square step grid size (" << gridSizeSquare << ", " << gridSizeSquare<< ");\n";
	SquareStepParallel<<<gridDimension, blockDimension>>> (dev_Map, dev_Randoms, size, step, randomScale);
}

void DiamondSquareParallel::CleanUp ()
{
	CHECK (hipFree(dev_Randoms))
	CHECK (hipFree(dev_Map))
}

