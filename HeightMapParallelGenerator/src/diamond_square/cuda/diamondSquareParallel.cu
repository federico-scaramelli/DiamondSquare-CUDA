#include "hip/hip_runtime.h"
﻿#include "diamondSquareParallel.h"
#include "../parameters/applicationSettings.h"
#include "hip/hip_runtime.h"
#include ""

__device__ __forceinline__ uint32_t GetIndexOnDevice (uint32_t x, uint32_t y, uint32_t size)
{
	x = x >= size ? size - 1 : x;
	y = y >= size ? size - 1 : y;

	return x * size + y;
}


#pragma region Random Generator

#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

inline bool GetRandomOnHost (float* const value)
{
	bool cond = static_cast<int> (*value * 10) & 0x01;
	*value = *value * (-1) * cond + *value * !cond;
	return cond;
}

void DiamondSquareParallel::PrintRandoms ()
{
	map = new float[totalSize];
	CHECK (hipMemcpy(map, dev_Map, totalSize * sizeof(float), hipMemcpyDeviceToHost))
	auto count = 0;

	for (int i = 0; i < totalSize; i++) {
		count = GetRandomOnHost (map + i) ? count + 1 : count;
		//std::cout << randoms[i];
	}
	std::cout << count << " negativi" << std::endl;
	std::cout << totalSize - count << " positivi" << std::endl;
}

void DiamondSquareParallel::GenerateRandomNumbers_HostAPI ()
{
	int seed = RandomIntUniform();
	hiprandGenerator_t generator;
	CHECK_CURAND (hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MT19937))
	CHECK_CURAND (hiprandSetGeneratorOrdering(generator, HIPRAND_ORDERING_PSEUDO_BEST));
	CHECK_CURAND (hiprandSetPseudoRandomGeneratorSeed(generator, seed))

	/* Generate n floats on device */
	CHECK_CURAND (hiprandGenerateUniform(generator, dev_Map, totalSize))

	//PrintRandoms();

	/* Cleanup */
	CHECK_CURAND (hiprandDestroyGenerator(generator))
}

__global__ void SetupRandomGenerator (hiprandStateMRG32k3a* state, int n, int totalSize, int seed)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx * n > totalSize) return;

	/* Each thread gets same seed, a different sequence
	   number, no offset */
	hiprand_init (seed + idx, 0, 0, &state[idx]);
}

__global__ void GenerateRandomNumbers (float* map, hiprandStateMRG32k3a* state, int n, int totalSize)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	float x;
	/* Copy state to local memory for efficiency */
	hiprandStateMRG32k3a localState = state[idx];

	/* Generate pseudo-random uniforms */
	for (int i = 0; i < n; i++) {
		if (idx * n + i > totalSize) return;
		x = hiprand_uniform_double (&localState);
		map[idx * n + i] = x;
	}
	/* Copy state back to global memory */
	state[idx] = localState;
}

void DiamondSquareParallel::GenerateRandomNumbers_DeviceAPI ()
{
	int n = 128;

	CHECK (hipMalloc((void **)&dev_MRGStates, (totalSize + n - 1) / n * sizeof(hiprandStateMRG32k3a)))

	dim3 blockSize (MAX_BLOCK_SIZE * MAX_BLOCK_SIZE, 1, 1);
	dim3 gridSize ((((totalSize + n - 1) / n) + (blockSize.x - 1)) / blockSize.x, 1, 1);
	SetupRandomGenerator<<<gridSize, blockSize>>> (dev_MRGStates, n, totalSize, RandomIntUniform());
	hipDeviceSynchronize();
	GenerateRandomNumbers<<<gridSize, blockSize>>> (dev_Map, dev_MRGStates, n, totalSize);
	hipDeviceSynchronize();

	/*CHECK (hipMemcpy(map, dev_Map, totalSize * sizeof(float), hipMemcpyDeviceToHost))
	PrintMap();*/
}

__device__ __forceinline__ float GetRandomOnDevice (float const value)
{
	bool cond = static_cast<int> (value * 10) & 0x01;
	return value * (-1) * cond + value * !cond;
}

#pragma endregion


#pragma region Initialization

void DiamondSquareParallel::AllocateMapOnDevice()
{
	CHECK (hipMalloc ((void**)&dev_Map, totalSize * sizeof(float)))
}

void DiamondSquareParallel::InitializeDiamondSquare ()
{
	std::cout << "================== PARALLEL DIAMOND SQUARE ==================\n\n";
	std::cout << "---------- INITIALIZATION ----------\n";
	std::cout << "Initializing Diamond Square [" << size << " x " << size << "]...\n";

	MeasureTimeFn (nullptr, "Allocation time on device is ", this, &DiamondSquareParallel::AllocateMapOnDevice);

	threadAmount = (size - 1) / step;

#if !CURAND_DEVICE
	MeasureTimeFn (nullptr, "Random number set with host API generated in ", this, 
	  &DiamondSquareParallel::GenerateRandomNumbers_HostAPI);
#else
	MeasureTimeFn (nullptr, "Random number set with device API generated in ", this,
	               &DiamondSquareParallel::GenerateRandomNumbers_DeviceAPI);
#endif
}

#pragma endregion


#pragma region Execution

void DiamondSquareParallel::ComputeBlockGridSizes ()
{
	/*			  2^k			  or			  MAX_BLOCK_SIZE			  */
	blockSizeDiamond = threadAmount <= MAX_BLOCK_SIZE ? threadAmount : MAX_BLOCK_SIZE;
	/*		(2^k + 1) x 2^(k+1)	  or	SQUARE_BLOCK_X_SIZE x MAX_BLOCK_SIZE
	*		        k <= 3					     k > 3						  */
	blockXSizeSquare = threadAmount <= SQUARE_BLOCK_X_SIZE ? blockSizeDiamond + 1 : SQUARE_BLOCK_X_SIZE;
	blockYSizeSquare = threadAmount <= SQUARE_BLOCK_X_SIZE ? threadAmount * 2 : blockSizeDiamond;

	/*				  1			  or			2^k / MAX_BLOCK_SIZE		  */
	gridSizeDiamond = (threadAmount + MAX_BLOCK_SIZE - 1) / MAX_BLOCK_SIZE;
	/* SQUARE_BLOCK_X_SIZE x MAX_BLOCK_SIZE			block amount
	 * = (2^(k+1) / MAX_BLOCK_SIZE)  x	 (2^k / SQUARE_BLOCK_X_SIZE) + 1	  */
	gridSizeXSquare = threadAmount <= SQUARE_BLOCK_X_SIZE ? 1 : (threadAmount / SQUARE_BLOCK_X_SIZE) + 1;
	gridSizeYSquare = (threadAmount * 2 + MAX_BLOCK_SIZE - 1) / MAX_BLOCK_SIZE;
}

void DiamondSquareParallel::DiamondSquare ()
{
#if CUDA_EVENTS_TIMING
	hipEvent_t start, stop;
	CHECK (hipEventCreate(&start))
	CHECK (hipEventCreate(&stop))
	CHECK (hipEventRecord( start, 0 ))
#endif

	while (step > 1) {
		ComputeBlockGridSizes();

		DiamondStep();
		CHECK (hipDeviceSynchronize())

#if PRINT_DIAMOND_STEP_CUDA
		CHECK (hipMemcpy(map, dev_Map, totalSize * sizeof(float), hipMemcpyDeviceToHost))
		PrintMap();
#endif

		SquareStep();
		CHECK (hipDeviceSynchronize())

#if PRINT_SQUARE_STEP_CUDA
		CHECK (hipMemcpy(map, dev_Map, totalSize * sizeof(float), hipMemcpyDeviceToHost))
		PrintMap();
#endif

		randomScale /= 2.0f;
		step /= 2;

		/* 2^k */
		threadAmount *= 2;
	}

#if COPY_RESULT_ON_HOST
	CHECK (hipMemcpy(map, dev_Map, totalSize * sizeof(float), hipMemcpyDeviceToHost))
#endif

	CleanUp();

#if CUDA_EVENTS_TIMING
	CHECK (hipEventRecord( stop, 0 ))
	CHECK (hipEventSynchronize( stop ))

	CHECK (hipEventElapsedTime( &executionTimeCuda, start, stop ))
	CHECK (hipEventDestroy( start ))
	CHECK (hipEventDestroy( stop ))
#endif
}

__global__ void DiamondStepParallel (float* map, uint32_t size, uint32_t step, float randomScale)
{
	uint32_t x = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t y = blockIdx.x * blockDim.x + threadIdx.x;
	x = x * step + (step / 2);
	y = y * step + (step / 2);

	float val = map[GetIndexOnDevice (x - (step / 2), y - (step / 2), size)] +
		map[GetIndexOnDevice (x + (step / 2), y - (step / 2), size)] +
		map[GetIndexOnDevice (x - (step / 2), y + (step / 2), size)] +
		map[GetIndexOnDevice (x + (step / 2), y + (step / 2), size)];

	val /= 4.0f;

	//map[GetIndexOnDeviceOnDevice (x, y, size)] = GetRandomOnDevice(map[GetIndexOnDeviceOnDevice (x, y, size)]) * randomScale + val;
	map[GetIndexOnDevice (x, y, size)] = (-1.0f + map[GetIndexOnDevice (x, y, size)] * 2.0f) * randomScale + val;
}

__global__ void SquareStepParallel (float* map, uint32_t size, uint32_t step, float randomScale)
{
	uint32_t thd_X = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

	uint32_t x = thd_X * step * (y % 2 == 0) +
		y * (step / 2) * (y % 2 != 0);
	y = (y * (step / 2) + (step / 2)) * (y % 2 == 0) +
		thd_X * step * (y % 2 != 0);

	if (x > size || y > size) {
		return;
	}

	float val = map[GetIndexOnDevice (x - (step / 2), y, size)] +
		map[GetIndexOnDevice (x + (step / 2), y, size)] +
		map[GetIndexOnDevice (x, y - (step / 2), size)] +
		map[GetIndexOnDevice (x, y + (step / 2), size)];

	val /= 4.0f;

	//map[GetIndexOnDeviceOnDevice (x, y, size)] = GetRandomOnDevice(map[GetIndexOnDeviceOnDevice (x, y, size)]) * randomScale + val;
	map[GetIndexOnDevice (x, y, size)] = (-1.0f + map[GetIndexOnDevice (x, y, size)] * 2.0f) * randomScale + val;
}

void DiamondSquareParallel::DiamondStep ()
{
	dim3 blockDimension (blockSizeDiamond, blockSizeDiamond, 1);
	dim3 gridDimension (gridSizeDiamond, gridSizeDiamond, 1);

	DiamondStepParallel<<<gridDimension, blockDimension>>> (dev_Map, size, step, randomScale);
}

void DiamondSquareParallel::SquareStep ()
{
	dim3 blockDimension (blockXSizeSquare, blockYSizeSquare, 1);
	dim3 gridDimension (gridSizeXSquare, gridSizeYSquare, 1);

	SquareStepParallel<<<gridDimension, blockDimension>>> (dev_Map, size, step, randomScale);
}

#pragma endregion


#pragma region Values Mapping

//__device__ __forceinline__ float atomicMinFloat(float* addr, float value) {
//    // Source: https://stackoverflow.com/questions/17399119/how-do-i-use-atomicmax-on-floating-point-values-in-cuda/51549250#51549250
//    float old;
//    old = (value >= 0) ? __int_as_float(atomicMin((int*)addr, __float_as_int(value))) :
//        __uint_as_float(atomicMax((unsigned int*)addr, __float_as_uint(value)));
//    return old;
//}

void DiamondSquareParallel::MapValuesToGrayScale ()
{
	grayScaleMap = new uint8_t[totalSize]{ 0 };
	auto minmax = std::minmax_element (map, map + totalSize);

	for (uint32_t i = 0; i < size; ++i) {
		for (uint32_t j = 0; j < size; ++j) {
			grayScaleMap[i * size + j] = static_cast<uint8_t> (MapValue
				(*minmax.first, *minmax.second,
				 0, 255, map[i * size + j]));
		}
	}

	if (DELETE_DOUBLE_MAP)
		DeleteDoubleMap();
}

void DiamondSquareParallel::MapValuesToIntRange (int toMin, int toMax, int* outputMap)
{
	std::cout << "\n---------- VALUES MAPPING ----------" << std::endl;
	std::cout << "Mapping values..." << std::endl;

	auto minmax = std::minmax_element (map, map + totalSize);

	for (uint32_t i = 0; i < size; ++i) {
		for (uint32_t j = 0; j < size; ++j) {
			outputMap[i * size + j] = (MapValue
				(*minmax.first, *minmax.second,
				 toMin, toMax, map[i * size + j]));
		}
	}
}

#pragma endregion 

void DiamondSquareParallel::CleanUp ()
{
	CHECK (hipFree(dev_Map))
#if CURAND_DEVICE
	CHECK (hipFree(dev_MRGStates))
#endif
}
